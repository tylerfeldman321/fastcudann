//Based on the work of Andrew Krepps

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>
#include <random>
#include <iostream>
#include <chrono>
#include <algorithm>
using namespace std;

#define ARRAY_SIZE (1 << 25)
#define ARRAY_SIZE_IN_BYTES (sizeof(int) * (ARRAY_SIZE))
#define NUM_STREAMS 8

inline hipError_t checkCuda(hipError_t result)
{
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
  return result;
}

__global__
void work_kernel(int *a, int* result, int N)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = tid; i < N; i += stride) {
		result[i] = a[i] + 100;
	}
}


int main(int argc, char** argv)
{
	int totalThreads = (1 << 20);
	int blockSize = 256;  // Also threads / block
	int N = 10;

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// Allocate and initialize memory
	int* a_cpu;
	int* result_cpu;
	a_cpu = (int*)malloc(N * sizeof(int));
	result_cpu = (int*)malloc(N * sizeof(int));

	for (int i = 0; i < N; i++) {
		a_cpu[i] = 1;
	}

	int* a_gpu;
	int* result_gpu;
	checkCuda( hipMalloc((void **)&a_gpu, N*sizeof(int)) );
	checkCuda( hipMalloc((void **)&result_gpu, N*sizeof(int)) );
	hipMemcpy( a_gpu, a_cpu, N*sizeof(int), hipMemcpyHostToDevice );

	hipEventRecord(start);
	work_kernel<<<1, 1>>>(a_gpu, result_gpu, N);
	hipEventRecord(stop);

	// Copy back results, synchronize
	checkCuda( hipMemcpy( result_cpu, result_gpu, N*sizeof(int), hipMemcpyDeviceToHost ) );
	checkCuda( hipDeviceSynchronize() );
	if (true) {
		printf("Results of operation: \n");
		for (int i = 0; i < N; i++) {
			printf("Result[%d]: %d\n", i, result_cpu[i]);
		}
	}

	// Print runtime information
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("Milliseconds elapsed: %f\n", milliseconds);

	// Free memory
	checkCuda( hipFree(a_gpu) );
	checkCuda( hipFree(result_gpu) );
	free(a_cpu);
	free(result_cpu);
}
