#include <iostream>
#include "../include/mnist_reader_common.hpp"
#include "../include/utils.cuh"
#include "../include/ops.cuh"


#define WARMUP_EPOCHS 5  // Number of warmup epochs (not counted in timing)


void print_mnist_image(const uint8_t* image_data, int rows, int columns, uint8_t label) {
    std::cout << "Label: " << static_cast<int>(label) << std::endl;
    for (int r = 0; r < rows; ++r) {
        for (int c = 0; c < columns; ++c) {
            uint8_t pixel = image_data[r * columns + c];
            std::cout << (pixel > 128 ? '#' : ' ');
        }
        std::cout << std::endl;
    }
    std::cout << "------------------------------" << std::endl;
}


bool run_basic_implementation(float *d_train_images, uint8_t *d_train_labels) {
    // Run my implementation w/o graphs
    size_t input_size = 784;
    size_t output_size = 10;
    size_t batch_size = 60000;
    float *h_weights = (float*)calloc(input_size*output_size, sizeof(float));

    int num_weights = input_size * output_size;
    size_t weights_bytes = sizeof(float)*input_size*output_size;
    size_t output_bytes = sizeof(float)*batch_size*output_size;
    size_t loss_bytes = sizeof(float)*batch_size;

    float *h_losses = (float*)malloc(loss_bytes);

    float learning_rate = 0.5;

    float *d_weights, *d_output, *d_probabilities, *d_losses, *d_grad_logits, *d_grad_weights;
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_weights, weights_bytes));
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_output, output_bytes));
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_probabilities, output_bytes));
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_losses, loss_bytes));
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_grad_logits, output_bytes));
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_grad_weights, weights_bytes));

    // Initialize weights
    init_weights_uniform<<<1, 1>>>(d_weights, input_size*output_size, 0);

    // Run forward and backward pass
    dim3 gridSize(1, 1);
    dim3 blockSize(1, 1);
    int num_epochs = 100;

    // --- Training Loop ---
    printf("Starting training for %d epochs...\n", num_epochs);
    for (int epoch = 0; epoch < num_epochs; ++epoch) {
        printf("Epoch: %d\n", epoch);

        // --- Forward Pass ---
        // 1. Calculate Logits
        matmul_kernel<<<gridSize, blockSize>>>(d_output, d_train_images, d_weights, input_size, output_size, batch_size);

        // 2. Calculate Probabilities
        softmax<<<1, 1>>>(d_output, d_probabilities, batch_size, output_size);
        
        // 3. Calculate Loss (per sample)
        scce_loss_forward_kernel<<<1, 1>>>(d_probabilities, d_train_labels, d_losses, batch_size, output_size);

        // --- Loss Calculation & Logging ---
        CHECK_CUDA_ERROR(hipMemcpy(h_losses, d_losses, loss_bytes, hipMemcpyDeviceToHost));
        CHECK_CUDA_ERROR(hipDeviceSynchronize());
        double total_loss = 0.0;
        for (int i = 0; i < batch_size; ++i) {
            total_loss += h_losses[i];
        }
        float average_loss = (float)(total_loss / batch_size);
        if ((epoch + 1) % 1 == 0 || epoch == 0 || epoch == num_epochs - 1) {
           printf("Epoch [%d/%d], Average Loss: %f\n", epoch + 1, num_epochs, average_loss);
        }


        // --- Backward Pass ---
        // 4. Calculate Gradient of Loss w.r.t. Logits (dL/dZ)
        scce_softmax_backward_kernel<<<1, 1>>>(d_probabilities, d_train_labels, d_grad_logits, batch_size, output_size);

        // 5. Calculate Gradient of Loss w.r.t Weights (dL/dW)
        calculate_weight_gradient_kernel<<<gridSize, blockSize>>>(d_grad_weights, d_train_images, d_grad_logits, input_size, output_size, batch_size);

        // --- Update Weights ---
        // 6. Apply gradient descent step
        update_weights_kernel<<<1, 1>>>(d_weights, d_grad_weights, learning_rate, num_weights);

        CHECK_CUDA_ERROR(hipGetLastError());
    }

    printf("Training finished.\n");
    hipDeviceSynchronize();

    CHECK_CUDA_ERROR(hipMemcpy(h_weights, d_weights, sizeof(float)*input_size*output_size, hipMemcpyDeviceToHost));
    CHECK_CUDA_ERROR( hipDeviceSynchronize());
    std::cout << "Weights: ";
    for (int i = 0; i < 10; i++)
        std::cout << h_weights[i] << " ";
    std::cout << std::endl;
    free(h_weights);

    float* h_output = (float*)malloc(batch_size*output_size*sizeof(float));
    CHECK_CUDA_ERROR(hipMemcpy(h_output, d_output, sizeof(float)*batch_size*output_size, hipMemcpyDeviceToHost));
    CHECK_CUDA_ERROR( hipDeviceSynchronize());
    std::cout << "Logits: ";
    for (int i = 0; i < 10; i++)
        std::cout << h_output[i] << " ";
    std::cout << std::endl;
    free(h_output);

    float* h_prob = (float*)malloc(batch_size*output_size*sizeof(float));
    CHECK_CUDA_ERROR(hipMemcpy(h_prob, d_probabilities, sizeof(float)*batch_size*output_size, hipMemcpyDeviceToHost));
    CHECK_CUDA_ERROR(hipDeviceSynchronize());
    std::cout << "Probabilities: ";
    float sum = 0;
    for (int i = 0; i < 10; i++){
        std::cout << h_prob[i] << " ";
        sum += h_prob[i];}
    std::cout << std::endl;
    std::cout << sum << std::endl;
    free(h_prob);

    CHECK_CUDA_ERROR(hipFree(d_weights));
    CHECK_CUDA_ERROR(hipFree(d_output));
    CHECK_CUDA_ERROR(hipFree(d_probabilities));
    CHECK_CUDA_ERROR(hipFree(d_losses));
    CHECK_CUDA_ERROR(hipFree(d_grad_logits));
    CHECK_CUDA_ERROR(hipFree(d_grad_weights));

    return true;
}


int main(int argc, char* argv[]) {
    std::string MNIST_DATA_LOCATION = "/home/ubuntu/fastcudann/data";
    std::cout << "MNIST data directory: " << MNIST_DATA_LOCATION << std::endl;
    
    // Read train images
    auto mnist_train_data_buffer = read_mnist_file(MNIST_DATA_LOCATION + "/train-images.idx3-ubyte", 0x803);
    int train_images_count   = static_cast<int>(read_header(mnist_train_data_buffer, 1));
    int train_images_rows    = static_cast<int>(read_header(mnist_train_data_buffer, 2));
    int train_images_columns = static_cast<int>(read_header(mnist_train_data_buffer, 3));
    uint8_t* train_images = reinterpret_cast<uint8_t*>(mnist_train_data_buffer.get() + 16);
    std::cout << train_images_count << std::endl;

    // Read train labels
    auto mnist_train_labels_data_buffer = read_mnist_file(MNIST_DATA_LOCATION + "/train-labels.idx1-ubyte", 0x801);
    auto train_labels_count = read_header(mnist_train_labels_data_buffer, 1);
    auto train_labels = reinterpret_cast<uint8_t*>(mnist_train_labels_data_buffer.get() + 8);
    std::cout << train_labels_count << std::endl;

    // Read test images
    auto mnist_test_data_buffer = read_mnist_file(MNIST_DATA_LOCATION + "/t10k-images.idx3-ubyte", 0x803);
    int test_images_count   = static_cast<int>(read_header(mnist_test_data_buffer, 1));
    int test_images_rows    = static_cast<int>(read_header(mnist_test_data_buffer, 2));
    int test_images_columns = static_cast<int>(read_header(mnist_test_data_buffer, 3));
    uint8_t* test_images = reinterpret_cast<uint8_t*>(mnist_test_data_buffer.get() + 16);
    std::cout << test_images_count << std::endl;

    // Read test labels
    auto mnist_test_labels_data_buffer = read_mnist_file(MNIST_DATA_LOCATION + "/t10k-labels.idx1-ubyte", 0x801);
    auto test_labels_count = read_header(mnist_test_labels_data_buffer, 1);
    auto test_labels = reinterpret_cast<uint8_t*>(mnist_test_labels_data_buffer.get() + 8);
    std::cout << test_labels_count << std::endl;

    // // --- Print Samples ---
    // std::cout << "\n--- Printing Training Samples ---" << std::endl;
    // int image_size = train_images_rows * train_images_columns; // Should be 784 for MNIST
    // print_mnist_image(train_images + 0 * image_size, train_images_rows, train_images_columns, train_labels[0]);
    // print_mnist_image(train_images + 1 * image_size, train_images_rows, train_images_columns, train_labels[1]);
    // std::cout << "\n--- Printing Testing Samples ---" << std::endl;
    // int test_image_size = test_images_rows * test_images_columns;
    // print_mnist_image(test_images + 0 * test_image_size, test_images_rows, test_images_columns, test_labels[0]);
    // print_mnist_image(test_images + 1 * test_image_size, test_images_rows, test_images_columns, test_labels[1]);

    uint8_t *d_train_images_uint8, *d_train_labels;
    size_t num_training_pixels = (size_t)train_images_count*train_images_rows*train_images_columns;
    size_t training_labels_size = sizeof(uint8_t)*train_labels_count;
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_train_images_uint8, sizeof(uint8_t)*num_training_pixels));
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_train_labels, training_labels_size));

    CHECK_CUDA_ERROR(hipMemcpy(d_train_images_uint8, train_images, sizeof(uint8_t)*num_training_pixels, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_train_labels, train_labels, training_labels_size, hipMemcpyHostToDevice));

    // Convert uint8_t data to normalized floats
    float *d_train_images_float;
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_train_images_float, sizeof(float) * num_training_pixels));
    convert_and_normalize<<<256, 256>>>(d_train_images_uint8, d_train_images_float, num_training_pixels);
    CHECK_CUDA_ERROR(hipFree(d_train_images_uint8));

    run_basic_implementation(d_train_images_float, d_train_labels);
        
    // TODO: Run my implementation with graphs
    // TODO: Run my implementation with graphs, reduce synchronization
    // TODO: Run cudNN

    CHECK_CUDA_ERROR(hipFree(d_train_images_float));
    CHECK_CUDA_ERROR(hipFree(d_train_labels));

    return 0;
}