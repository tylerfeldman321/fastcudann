#include "hip/hip_runtime.h"
#include "../include/train.cuh"
#include <hip/hip_runtime.h>
#include "../include/utils.cuh"
#include "../include/ops.cuh"

// Function to calculate accuracy on the device (avoids transferring probabilities)
__global__ void calculate_accuracy_kernel(const float* probabilities, const uint8_t* true_labels,
                                          int* correct_counts,
                                          int batch_size, int num_classes) {
    size_t thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
    size_t stride = blockDim.x * gridDim.x;

    for (size_t sample_idx = thread_idx; sample_idx < batch_size; sample_idx += stride) {
        int offset = sample_idx * num_classes;
        const float* current_probs = probabilities + offset;
        int true_label = (int)true_labels[sample_idx];

        // Find the index of the highest probability (predicted class)
        int predicted_label = 0;
        float max_prob = current_probs[0];
        for (int j = 1; j < num_classes; ++j) {
            if (current_probs[j] > max_prob) {
                max_prob = current_probs[j];
                predicted_label = j;
            }
        }

        // If prediction matches true label, atomically increment the counter
        if (predicted_label == true_label) {
            atomicAdd(correct_counts, 1); // Atomically add 1 to the shared counter
        }
    }
}


bool run_training_basic_implementation(float *d_all_train_images_float, // Pointer to ALL training images on device
                  uint8_t *d_all_train_labels,     // Pointer to ALL training labels on device
                  int total_train_samples,         // Total number of training samples (e.g., 60000)
                  int input_size,                  // Size of one input image (e.g., 784)
                  int output_size,                 // Number of output classes (e.g., 10)
                  int num_epochs,                  // Number of epochs to train
                  int mini_batch_size,             // Mini-batch size
                  float learning_rate              // Learning rate for optimizer
                  )
{
    printf("Starting training...\n");
    printf("Parameters:\n");
    printf("  Epochs: %d\n", num_epochs);
    printf("  Mini-batch Size: %d\n", mini_batch_size);
    printf("  Learning Rate: %f\n", learning_rate);
    printf("  Input Size: %d\n", input_size);
    printf("  Output Size (Classes): %d\n", output_size);
    printf("  Total Training Samples: %d\n", total_train_samples);

    // --- Timing Setup ---
    hipEvent_t epoch_start_event, epoch_stop_event;
    CHECK_CUDA_ERROR(hipEventCreate(&epoch_start_event));
    CHECK_CUDA_ERROR(hipEventCreate(&epoch_stop_event));
    float epoch_gpu_time_ms = 0.0f;

    int num_weights = input_size * output_size;
    size_t weights_bytes = sizeof(float) * num_weights;
    // Allocate intermediate buffers based on MINI_BATCH_SIZE
    size_t output_bytes = sizeof(float) * mini_batch_size * output_size;
    size_t loss_bytes = sizeof(float) * mini_batch_size;
    size_t accuracy_counter_bytes = sizeof(int); // For single atomic counter

    float *h_losses = (float*)malloc(loss_bytes); // Host buffer for losses of one mini-batch
    if (!h_losses) { fprintf(stderr, "Failed to allocate host memory for losses\n"); return false; }
    int h_correct_count = 0; // Host variable for accuracy count

    // --- Device Memory Allocation ---
    float *d_weights, *d_output, *d_probabilities, *d_losses, *d_grad_logits, *d_grad_weights;
    int *d_correct_count;

    CHECK_CUDA_ERROR(hipMalloc((void**)&d_weights, weights_bytes));
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_grad_weights, weights_bytes));
    // Allocate based on mini_batch_size for intermediate results
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_output, output_bytes));
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_probabilities, output_bytes));
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_losses, loss_bytes));
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_grad_logits, output_bytes));
    // Allocate and initialize accuracy counter on device
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_correct_count, accuracy_counter_bytes));

    // --- Initialize Weights ---
    int init_grid_size = calculate_grid_size_1d(num_weights, BLOCK_SIZE_1D);
    init_weights_uniform<<<init_grid_size, BLOCK_SIZE_1D>>>(d_weights, num_weights, time(0));
    CHECK_CUDA_ERROR(hipGetLastError());

    // --- Training Loop ---
    int num_batches = (total_train_samples + mini_batch_size - 1) / mini_batch_size;
    printf("Total mini-batches per epoch: %d\n", num_batches);

    // Define block sizes
    dim3 block_1d(BLOCK_SIZE_1D);
    dim3 block_2d(BLOCK_DIM_2D, BLOCK_DIM_2D);

    for (int epoch = 0; epoch < num_epochs; ++epoch) {

        CHECK_CUDA_ERROR(hipEventRecord(epoch_start_event, 0));

        double epoch_total_loss = 0.0;
        long long epoch_total_correct = 0;
        long long epoch_total_processed = 0;

        for (int batch_idx = 0; batch_idx < num_batches; ++batch_idx) {
            int batch_start_idx = batch_idx * mini_batch_size;
            int current_batch_size = (batch_start_idx + mini_batch_size > total_train_samples) ?
                                      (total_train_samples - batch_start_idx) :
                                      mini_batch_size;

            if (current_batch_size <= 0) continue;
            float* d_current_batch_images = d_all_train_images_float + batch_start_idx * input_size;
            uint8_t* d_current_batch_labels = d_all_train_labels + batch_start_idx;


            // --- Forward Pass ---
            // 1. Calculate Logits (d_output = d_current_batch_images * d_weights)
            dim3 matmul_grid = calculate_grid_size_2d(current_batch_size, output_size, block_2d);
            matmul_kernel<<<matmul_grid, block_2d>>>(d_output, d_current_batch_images, d_weights, input_size, output_size, current_batch_size);

            // 2. Calculate Probabilities (d_probabilities = softmax(d_output))
            int softmax_grid = calculate_grid_size_1d(current_batch_size, BLOCK_SIZE_1D);
            softmax<<<softmax_grid, block_1d>>>(d_output, d_probabilities, current_batch_size, output_size);

            // 3. Calculate Loss (per sample for the current batch)
            int loss_grid = calculate_grid_size_1d(current_batch_size, BLOCK_SIZE_1D);
            scce_loss_forward_kernel<<<loss_grid, block_1d>>>(d_probabilities, d_current_batch_labels, d_losses, current_batch_size, output_size);


            // --- Loss Calculation & Logging ---
            CHECK_CUDA_ERROR(hipMemcpy(h_losses, d_losses, sizeof(float) * current_batch_size, hipMemcpyDeviceToHost));
            CHECK_CUDA_ERROR(hipDeviceSynchronize());

            // Sum losses on the CPU for this batch
            double current_batch_total_loss = 0.0;
            for (int i = 0; i < current_batch_size; ++i) {
                current_batch_total_loss += h_losses[i];
            }
            epoch_total_loss += current_batch_total_loss;
            epoch_total_processed += current_batch_size;


             // --- Calculate Accuracy (on GPU) ---
            CHECK_CUDA_ERROR(hipMemset(d_correct_count, 0, accuracy_counter_bytes)); // Reset counter for the batch
            int accuracy_grid = calculate_grid_size_1d(current_batch_size, BLOCK_SIZE_1D);
            calculate_accuracy_kernel<<<accuracy_grid, block_1d>>>(d_probabilities, d_current_batch_labels, d_correct_count, current_batch_size, output_size);
            // Copy the result back from GPU
            CHECK_CUDA_ERROR(hipMemcpy(&h_correct_count, d_correct_count, accuracy_counter_bytes, hipMemcpyDeviceToHost));
            CHECK_CUDA_ERROR(hipDeviceSynchronize());  // Synchronize ensure the accuracy count copy is complete.
            epoch_total_correct += h_correct_count;


            // --- Backward Pass ---
            // 4. Calculate Gradient of Loss w.r.t. Logits (dL/dZ)
            float grad_scale_factor = 1.0f / (float)current_batch_size; // Average gradient over the batch
            int backward_grid = calculate_grid_size_1d(current_batch_size, BLOCK_SIZE_1D); // Based on batch size
            scce_softmax_backward_kernel<<<backward_grid, block_1d>>>(d_probabilities, d_current_batch_labels, d_grad_logits, current_batch_size, output_size, grad_scale_factor);

            // 5. Calculate Gradient of Loss w.r.t Weights (dL/dW = X^T * dL/dZ)
            // Grid depends on weight matrix dimensions (input_size x output_size)
            dim3 grad_weights_grid = calculate_grid_size_2d(input_size, output_size, block_2d);
            calculate_weight_gradient_kernel<<<grad_weights_grid, block_2d>>>(d_grad_weights, d_current_batch_images, d_grad_logits, input_size, output_size, current_batch_size);

            // --- Update Weights ---
            // 6. Apply gradient descent step (Weights = Weights - LR * dL/dW)
            int update_grid = calculate_grid_size_1d(num_weights, BLOCK_SIZE_1D);
            update_weights_kernel<<<update_grid, block_1d>>>(d_weights, d_grad_weights, learning_rate, num_weights);

            // Check for errors periodically (e.g., end of batch) - essential for debugging
            CHECK_CUDA_ERROR(hipGetLastError());
        }

        CHECK_CUDA_ERROR(hipDeviceSynchronize());

        // --- Record Epoch Stop Time (GPU) and Calculate Duration ---
        CHECK_CUDA_ERROR(hipEventRecord(epoch_stop_event, 0));
        CHECK_CUDA_ERROR(hipEventSynchronize(epoch_stop_event));
        CHECK_CUDA_ERROR(hipEventElapsedTime(&epoch_gpu_time_ms, epoch_start_event, epoch_stop_event));

        // Calculate and log average loss and accuracy for the epoch
        float average_loss = (epoch_total_processed > 0) ? (float)(epoch_total_loss / epoch_total_processed) : 0.0f;
        float accuracy = (epoch_total_processed > 0) ? (float)(epoch_total_correct * 100.0 / epoch_total_processed) : 0.0f;

        printf("Epoch [%d/%d], Average Loss: %.6f, Accuracy: %.2f%%, Epoch GPU Time: %.2f ms (%.3f s)\n",
                epoch + 1, num_epochs, average_loss, accuracy, epoch_gpu_time_ms, epoch_gpu_time_ms / 1000.0f);
    }
    printf("Training complete!\n");

    // --- Cleanup ---
    free(h_losses);
    CHECK_CUDA_ERROR(hipFree(d_weights));
    CHECK_CUDA_ERROR(hipFree(d_output));
    CHECK_CUDA_ERROR(hipFree(d_probabilities));
    CHECK_CUDA_ERROR(hipFree(d_losses));
    CHECK_CUDA_ERROR(hipFree(d_grad_logits));
    CHECK_CUDA_ERROR(hipFree(d_grad_weights));
    CHECK_CUDA_ERROR(hipFree(d_correct_count));

    return true;
}


// --- Modified Training Function ---
bool run_training_optimized(
    float *d_all_train_images_float,
    uint8_t *d_all_train_labels,
    int total_train_samples,
    int input_size,
    int output_size,
    int num_epochs,
    int mini_batch_size,
    float learning_rate,
    int loss_print_period
) {
    printf("Starting training (Periodic Loss Reporting)...\n");
    printf("Parameters:\n");
    printf("  Epochs: %d\n", num_epochs);
    printf("  Mini-batch Size: %d\n", mini_batch_size);
    printf("  Learning Rate: %f\n", learning_rate);
    printf("  Input Size: %d\n", input_size);
    printf("  Output Size (Classes): %d\n", output_size);
    printf("  Total Training Samples: %d\n", total_train_samples);
    printf("  Loss Print Period: %d epochs\n", loss_print_period);

    // --- Timing Setup ---
    hipEvent_t epoch_start_event, epoch_stop_event;
    CHECK_CUDA_ERROR(hipEventCreate(&epoch_start_event));
    CHECK_CUDA_ERROR(hipEventCreate(&epoch_stop_event));
    float epoch_gpu_time_ms = 0.0f;

    // --- Memory Sizes ---
    int num_weights = input_size * output_size;
    size_t weights_bytes = sizeof(float) * num_weights;
    size_t output_bytes = sizeof(float) * mini_batch_size * output_size;
    size_t batch_loss_bytes = sizeof(float) * mini_batch_size;
    size_t scalar_float_bytes = sizeof(float);
    size_t scalar_int_bytes = sizeof(int);

    // --- Host Variables for Periodic Reporting ---
    float h_epoch_total_loss = 0.0f;
    int h_epoch_total_correct = 0;

    // --- Device Memory Allocation ---
    float *d_weights, *d_grad_weights;
    float *d_output, *d_probabilities, *d_grad_logits;
    float *d_batch_losses;
    float *d_epoch_total_loss;
    int   *d_epoch_total_correct;

    CHECK_CUDA_ERROR(hipMalloc((void**)&d_weights, weights_bytes));
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_grad_weights, weights_bytes));
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_output, output_bytes));
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_probabilities, output_bytes));
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_grad_logits, output_bytes));
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_batch_losses, batch_loss_bytes));
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_epoch_total_loss, scalar_float_bytes));
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_epoch_total_correct, scalar_int_bytes));

    // --- Initialize Weights ---
    int init_grid_size = calculate_grid_size_1d(num_weights, BLOCK_SIZE_1D);
    init_weights_uniform<<<init_grid_size, BLOCK_SIZE_1D>>>(d_weights, num_weights, time(0));
    CHECK_CUDA_ERROR(hipGetLastError());

    // --- Training Loop ---
    int num_batches = (total_train_samples + mini_batch_size - 1) / mini_batch_size;
    printf("Total mini-batches per epoch: %d\n", num_batches);

    // Define block sizes
    dim3 block_1d(BLOCK_SIZE_1D);
    dim3 block_2d(BLOCK_DIM_2D, BLOCK_DIM_2D);

    for (int epoch = 0; epoch < num_epochs; ++epoch) {

        CHECK_CUDA_ERROR(hipEventRecord(epoch_start_event, 0));

        // Reset GPU accumulators at the beginning of each epoch
        CHECK_CUDA_ERROR(hipMemsetAsync(d_epoch_total_loss, 0, scalar_float_bytes, 0));
        CHECK_CUDA_ERROR(hipMemsetAsync(d_epoch_total_correct, 0, scalar_int_bytes, 0));

        long long epoch_total_processed = 0; // Track samples processed on CPU side

        for (int batch_idx = 0; batch_idx < num_batches; ++batch_idx) {
            int batch_start_idx = batch_idx * mini_batch_size;
            int current_batch_size = (batch_start_idx + mini_batch_size > total_train_samples) ?
                                       (total_train_samples - batch_start_idx) :
                                       mini_batch_size;

            if (current_batch_size <= 0) continue;

            float* d_current_batch_images = d_all_train_images_float + batch_start_idx * input_size;
            uint8_t* d_current_batch_labels = d_all_train_labels + batch_start_idx;

            // --- Forward Pass ---
            // 1. Calculate Logits (d_output = d_current_batch_images * d_weights)
            dim3 matmul_grid = calculate_grid_size_2d(current_batch_size, output_size, block_2d);
            matmul_kernel<<<matmul_grid, block_2d>>>(d_output, d_current_batch_images, d_weights, input_size, output_size, current_batch_size);

            // 2. Calculate Probabilities (d_probabilities = softmax(d_output))
            int softmax_grid = calculate_grid_size_1d(current_batch_size, BLOCK_SIZE_1D);
            softmax<<<softmax_grid, block_1d>>>(d_output, d_probabilities, current_batch_size, output_size);

            // --- Loss and Accuracy Calculation (GPU Accumulation) ---
            // 3. Calculate Loss (per sample) and Accumulate total epoch loss on GPU
            int loss_grid = calculate_grid_size_1d(current_batch_size, BLOCK_SIZE_1D);
            // *** Requires kernel modification ***
            // Assumes kernel writes per-sample loss to d_batch_losses AND atomically adds the sum of d_batch_losses to d_epoch_total_loss
            scce_loss_forward_kernel_accumulate<<<loss_grid, block_1d>>>(
                d_probabilities, d_current_batch_labels, d_batch_losses, d_epoch_total_loss,
                current_batch_size, output_size);

            // 4. Calculate Accuracy and Accumulate total correct count on GPU
            int accuracy_grid = calculate_grid_size_1d(current_batch_size, BLOCK_SIZE_1D);
            // *** Requires kernel modification ***
            // Assumes kernel atomically increments d_epoch_total_correct
            calculate_accuracy_kernel_accumulate<<<accuracy_grid, block_1d>>>(
                d_probabilities, d_current_batch_labels, d_epoch_total_correct,
                current_batch_size, output_size);

            // --- Backward Pass ---
            // 5. Calculate Gradient of Loss w.r.t. Logits (dL/dZ)
            float grad_scale_factor = 1.0f / (float)current_batch_size;
            int backward_grid = calculate_grid_size_1d(current_batch_size, BLOCK_SIZE_1D);
            scce_softmax_backward_kernel<<<backward_grid, block_1d>>>(d_probabilities, d_current_batch_labels, d_grad_logits, current_batch_size, output_size, grad_scale_factor);

            // 6. Calculate Gradient of Loss w.r.t Weights (dL/dW = X^T * dL/dZ)
            dim3 grad_weights_grid = calculate_grid_size_2d(input_size, output_size, block_2d);
            calculate_weight_gradient_kernel<<<grad_weights_grid, block_2d>>>(d_grad_weights, d_current_batch_images, d_grad_logits, input_size, output_size, current_batch_size);

            // --- Update Weights ---
            // 7. Apply gradient descent step (Weights = Weights - LR * dL/dW)
            int update_grid = calculate_grid_size_1d(num_weights, BLOCK_SIZE_1D);
            update_weights_kernel<<<update_grid, block_1d>>>(d_weights, d_grad_weights, learning_rate, num_weights);

            // --- Update CPU counter for total processed samples ---
            epoch_total_processed += current_batch_size;
        }

        // Record Epoch Stop Time (GPU)
        CHECK_CUDA_ERROR(hipEventRecord(epoch_stop_event, 0));

        // --- Conditional Loss/Accuracy Reporting ---
        bool should_print_loss = ((epoch + 1) % loss_print_period == 0) || (epoch == num_epochs - 1);
        if (should_print_loss) {
            CHECK_CUDA_ERROR(hipEventSynchronize(epoch_stop_event));
            CHECK_CUDA_ERROR(hipDeviceSynchronize());

            // Copy accumulated results from GPU to Host
            CHECK_CUDA_ERROR(hipMemcpy(&h_epoch_total_loss, d_epoch_total_loss, scalar_float_bytes, hipMemcpyDeviceToHost));
            CHECK_CUDA_ERROR(hipMemcpy(&h_epoch_total_correct, d_epoch_total_correct, scalar_int_bytes, hipMemcpyDeviceToHost));

            CHECK_CUDA_ERROR(hipEventElapsedTime(&epoch_gpu_time_ms, epoch_start_event, epoch_stop_event));
            
            float average_loss = (epoch_total_processed > 0) ? (h_epoch_total_loss / epoch_total_processed) : 0.0f;
            float accuracy = (epoch_total_processed > 0) ? (float)(h_epoch_total_correct * 100.0 / epoch_total_processed) : 0.0f;

            printf("Epoch [%d/%d], Average Loss: %.6f, Accuracy: %.2f%%, Epoch GPU Time: %.2f ms (%.3f s)\n",
                   epoch + 1, num_epochs, average_loss, accuracy, epoch_gpu_time_ms, epoch_gpu_time_ms / 1000.0f);

        } else {
             CHECK_CUDA_ERROR(hipEventSynchronize(epoch_stop_event));
             CHECK_CUDA_ERROR(hipEventElapsedTime(&epoch_gpu_time_ms, epoch_start_event, epoch_stop_event));
        }
        CHECK_CUDA_ERROR(hipGetLastError());
    }

    printf("Training complete!\n");

    // --- Cleanup ---
    CHECK_CUDA_ERROR(hipEventDestroy(epoch_start_event));
    CHECK_CUDA_ERROR(hipEventDestroy(epoch_stop_event));

    CHECK_CUDA_ERROR(hipFree(d_weights));
    CHECK_CUDA_ERROR(hipFree(d_output));
    CHECK_CUDA_ERROR(hipFree(d_probabilities));
    CHECK_CUDA_ERROR(hipFree(d_batch_losses));
    CHECK_CUDA_ERROR(hipFree(d_grad_logits));
    CHECK_CUDA_ERROR(hipFree(d_grad_weights));
    CHECK_CUDA_ERROR(hipFree(d_epoch_total_loss));
    CHECK_CUDA_ERROR(hipFree(d_epoch_total_correct));

    return true;
}